#include "hip/hip_runtime.h"
#include "fft_module.cuh"
#include "cuda_module.h"
#include "hip/hip_runtime.h"
#include ""
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

#include <stdio.h>
#include <iostream>
#include <hipfft/hipfft.h>
#include <complex>

//int main(int argc, char **argv){}

//The cufft must be invoked by the host, not as part of a kernel. 
//num_wins = number of requested frames + averaging -1, this is required to ensure the correct averaging parameters. 
//a requested frame is an FFT of size = resolution
//by the same logic, h_samp_array must be = num_wins * resolution. h_out will be = (num_wins - (averaging-1)) * resolution
void perform_fft(std::complex<short>* h_samp_arry, float* h_out, const int resolution, const int averaging, const int num_wins) {
	/*
	if (num_wins == 0) {
		std::cout << "AMG NO WINS!\n";
		num_wins = sizeof(h_samp_arry) / (2 * resolution);
		std::cout << "Number of windows: " << num_wins << std::endl;
	} */
	//const int num_wins = 1;
	//hipComplex* samp[resolution];
	//std::complex<short>* d_samp;

	hipError_t cudaStatus;
	hipfftResult fftStatus;

	//Create cufft plan, turns out cufft handles its own memory transfers, so we must use callbacks in order to avoid numerous reads and writes in the device
	//Will however use multiple kernels initially, then see what the performance improvement is with callbacks at a later stage. n.n
	hipfftHandle plan;
	fftStatus = hipfftPlan1d(&plan, resolution, HIPFFT_C2C, (num_wins + averaging - 1)); //is deprecated
	//int n[1] = { resolution };
	//fftStatus = hipfftPlanMany(&plan, 1, n, 
	//	NULL, 1, resolution, 
	///	NULL, 1, resolution, 
	// HIPFFT_C2C, (num_wins + averaging - 1));
	if (fftStatus != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error creating plan: %d\n", fftStatus);
		goto Error;
	}

	// for outputting of averaged and processed samples /
	
	float* d_out;
	
	//cast stl complex to cuda complex
	cuComplexShort* h_samp_ptr = (cuComplexShort*)&h_samp_arry[0];

	//std::cout << h_samp_arry[0].real() << "," << h_samp_arry[0].imag() << " cuCmplx" << h_samp_ptr[0].x << "," << h_samp_ptr[0].y << std::endl;
	
	float* h_coef;
	h_coef = (float*)malloc(sizeof(float)*resolution);
	float* d_coef;

	cuComplexShort* d_samp;
	hipComplex* d_fftbuff;

	float win_power = 0;
	int rx_gain = 30;

	//Create coefficient array and x axis index for plotting
	for (int i = 0; i < resolution; i++) {
		h_coef[i] = 0.35875 - 0.48829*cos(2 * pi*i / (resolution - 1)) + 0.14128*cos(4 * pi*i / (resolution - 1)) - 0.01168*cos(6 * pi*i / (resolution - 1)); //blackmann harris window		
		win_power += (h_coef[i] * h_coef[i]); //this computes the total window power and normalises it to account for DC gain due to the window.
	}
	win_power /= resolution; //normalise the total window power across each sample.

	const float offset = 10 - rx_gain + 10 * std::log10(win_power); //10 is the MAX power detected by the ADC and take into account the gain of the frontend.

	//printf("GPU Offset: %f", offset);

	cuda_memcheck();



	//allocate the memory for the GPU
	cudaStatus = hipMalloc((cuComplexShort**)&d_samp, sizeof(cuComplexShort)* resolution*(num_wins + averaging - 1));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "d_samp hipMalloc failed! %s", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cuda_memcheck();
	
	cudaStatus = hipMalloc((float**)&d_coef, sizeof(float)*resolution);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "d_coef hipMalloc failed! %s", hipGetErrorString(cudaStatus));
		goto Error;
	}
	
	cuda_memcheck();

	cudaStatus = hipMalloc((hipComplex**)&d_fftbuff, sizeof(hipComplex)*resolution*(num_wins + averaging - 1));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "d_fftbuff hipMalloc failed! %s", hipGetErrorString(cudaStatus));
		goto Error;
	}
	
	cuda_memcheck();

	//Transfer data to GPU
	cudaStatus = hipMemcpy(d_coef, h_coef, sizeof(float)*resolution, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy to Device failed! %s", hipGetErrorString(cudaStatus));
		goto Error;
	}
	
	cudaStatus = hipMemcpy(d_samp, h_samp_ptr, sizeof(cuComplexShort)*resolution*(num_wins + averaging - 1), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy to Device failed! %s", hipGetErrorString(cudaStatus));
		goto Error;
	}


	cufft_prep <<< (resolution*num_wins) / CU_THD, CU_THD >> > (d_fftbuff, d_samp, d_coef, (num_wins + averaging - 1), resolution); //This will create (WIN_SAMPS*num_wins)/CU_THD blocks, with 1024 threads per block
	
	checkCudaErrors(hipFree(d_samp));
	checkCudaErrors(hipFree(d_coef));

	//inplace fft
	fftStatus = hipfftExecC2C(plan, d_fftbuff, d_fftbuff, HIPFFT_FORWARD);
	if (fftStatus != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ExecC2C Forward failed %d\n", fftStatus);
		goto Error;
	}

	cudaStatus = hipMalloc((float**)&d_out, sizeof(float)*resolution * num_wins);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed! %s", hipGetErrorString(cudaStatus));
		goto Error;
	}
	hipMemset(d_out, 0, sizeof(float)*resolution * num_wins); //initialise to zero
	
	//Do something with the fft'd samples, like average them, then output them to the host, where the host can perform detection.
	avg_out <<< resolution / CU_THD, CU_THD >>> (d_out, d_fftbuff, num_wins, averaging, offset, resolution);
	filter <<< resolution / CU_THD, CU_THD >>> (d_out, num_wins, resolution); //As this uses the correct moving average, num_wins does not have to be divided out

	cudaStatus = hipMemcpy(h_out, d_out, sizeof(float)*resolution * num_wins, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy to Host failed! %s", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:

	hipfftDestroy(plan);
	checkCudaErrors(hipFree(d_out));
	//checkCudaErrors(hipFree(d_samp));
	//checkCudaErrors(hipFree(d_coef));
	checkCudaErrors(hipFree(d_fftbuff));

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed! %s", hipGetErrorString(cudaStatus));
	}

	//return h_out;
}

//Kernel Call
//https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-use-cufft-callbacks-custom-data-processing/ for inspiration
static __global__ void cufft_prep(hipComplex* d_fft, cuComplexShort* d_s, float* d_w, const int num_wins, const int resolution) {

	int idx = threadIdx.x;
	
	//blockDim = number of threads in a block
	//This will take an array of complex shorts (14b samples) an array of hipComplex and a window array, will convert the com_short to hipComplex (com_float), correctly scale the samples and apply the appropriate window prepping it for fft
	for (int i = blockIdx.x * blockDim.x + idx; i < resolution*num_wins; i += blockDim.x * gridDim.x){
		d_fft[i].x = (d_s[i].x*1.0f / 32767.0f) * d_w[i%resolution];
		d_fft[i].y = (d_s[i].y*1.0f / 32767.0f) * d_w[i%resolution];
	}
	//if(idx == 0) printf("d_s[%d]: %f,%f fftbuff %f,%f\n", idx, d_s[idx].x, d_s[idx].y, d_s[idx].x, d_s[idx].x);
}

static __global__ void filter(float*out, const int num_wins, const int resolution){

	int idx = threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	float* out_ptr = &out[0];
	const int fiveby_filter_level = 13; //normally 16 for 5x5, 13 for aggressive.
	const int filter_level = 5; // 3x3 kernel 

	bool FIVEBY = true; //for use later

	//increment loop by 1, and decrease total run by 1 to accomodate for edges of the kernel
	if (!FIVEBY){
		for (int i = (blockIdx.x * blockDim.x + idx) + stride; i < resolution*(num_wins - 1); i += stride){

			if (out_ptr[i] == 0 && (blockIdx.x + idx != 0 || blockIdx.x + idx != resolution - 1)){
				if ((out_ptr[i - 1 - stride] + out_ptr[i - stride] + out_ptr[i + 1 - stride] +
					out_ptr[i - 1] + out_ptr[i + 1] +
					out_ptr[i - 1 + stride] + out_ptr[i + stride] + out_ptr[i + 1 + stride]) > filter_level){

					out_ptr[i] = 1;
				}
			}
		}
	}
	if (FIVEBY){
		//special case code for handling the beginning and end of the image, note that edges are ignored as they are significantly less impactful on window generation

		for (int i = blockIdx.x * blockDim.x + idx, j = 0; i < stride*(num_wins); i += stride, j++){

			if (out_ptr[i] == 0 && (blockIdx.x * blockDim.x + idx != 0 && blockIdx.x * blockDim.x + idx != resolution - 1 &&
				blockIdx.x * blockDim.x + idx != 1 && blockIdx.x * blockDim.x + idx != resolution - 2)){

				if (j == 0){
					if (( //unrolled here for efficiencies
						out_ptr[i - 2] + out_ptr[i - 1] + out_ptr[i + 1] + out_ptr[i + 2] +
						out_ptr[i - 2 + stride] + out_ptr[i - 1 + stride] + out_ptr[i + stride] + out_ptr[i + 1 + stride] + out_ptr[i + 2 + stride] +
						out_ptr[i - 2 + 2 * stride] + out_ptr[i - 1 + 2 * stride] + out_ptr[i + 2 * stride] + out_ptr[i + 1 + 2 * stride] + out_ptr[i + 2 + 2 * stride]) > fiveby_filter_level - 6)
					{
						out_ptr[i] = 1;
					}
				}

				else if (j == 1){
					if (( //unrolled here for efficiencies
						out_ptr[i - 2 - stride] + out_ptr[i - 1 - stride] + out_ptr[i - stride] + out_ptr[i + 1 - stride] + out_ptr[i + 2 - stride] +
						out_ptr[i - 2] + out_ptr[i - 1] + out_ptr[i + 1] + out_ptr[i + 2] +
						out_ptr[i - 2 + stride] + out_ptr[i - 1 + stride] + out_ptr[i + stride] + out_ptr[i + 1 + stride] + out_ptr[i + 2 + stride] +
						out_ptr[i - 2 + 2 * stride] + out_ptr[i - 1 + 2 * stride] + out_ptr[i + 2 * stride] + out_ptr[i + 1 + 2 * stride] + out_ptr[i + 2 + 2 * stride]) > fiveby_filter_level - 3)
					{
						out_ptr[i] = 1;
					}
				}

				else if (j >= 2 && j < num_wins - 2){
					if (( //unrolled here for efficiencies
						out_ptr[i - 2 - 2 * stride] + out_ptr[i - 1 - 2 * stride] + out_ptr[i - 2 * stride] + out_ptr[i + 1 - 2 * stride] + out_ptr[i + 2 - 2 * stride] +
						out_ptr[i - 2 - stride] + out_ptr[i - 1 - stride] + out_ptr[i - stride] + out_ptr[i + 1 - stride] + out_ptr[i + 2 - stride] +
						out_ptr[i - 2] + out_ptr[i - 1] + out_ptr[i + 1] + out_ptr[i + 2] +
						out_ptr[i - 2 + stride] + out_ptr[i - 1 + stride] + out_ptr[i + stride] + out_ptr[i + 1 + stride] + out_ptr[i + 2 + stride] +
						out_ptr[i - 2 + 2 * stride] + out_ptr[i - 1 + 2 * stride] + out_ptr[i + 2 * stride] + out_ptr[i + 1 + 2 * stride] + out_ptr[i + 2 + 2 * stride]) > fiveby_filter_level)
					{
						out_ptr[i] = 1;
					}
				}

				else if (j == num_wins - 2){
					if (( //unrolled here for efficiencies, this isnt called ... bug
						out_ptr[i - 2 - 2 * stride] + out_ptr[i - 1 - 2 * stride] + out_ptr[i - 2 * stride] + out_ptr[i + 1 - 2 * stride] + out_ptr[i + 2 - 2 * stride] +
						out_ptr[i - 2 - stride] + out_ptr[i - 1 - stride] + out_ptr[i - stride] + out_ptr[i + 1 - stride] + out_ptr[i + 2 - stride] +
						out_ptr[i - 2] + out_ptr[i - 1] + out_ptr[i + 1] + out_ptr[i + 2] +
						out_ptr[i - 2 + stride] + out_ptr[i - 1 + stride] + out_ptr[i + stride] + out_ptr[i + 1 + stride] + out_ptr[i + 2 + stride]) > fiveby_filter_level - 3)
					{
						out_ptr[i] = 1;
					}
				}

				else if (j == num_wins - 1){
					if (( //unrolled here for efficiencies, neither is this ... BUG!!!!!
						out_ptr[i - 2 - 2 * stride] + out_ptr[i - 1 - 2 * stride] + out_ptr[i - 2 * stride] + out_ptr[i + 1 - 2 * stride] + out_ptr[i + 2 - 2 * stride] +
						out_ptr[i - 2 - stride] + out_ptr[i - 1 - stride] + out_ptr[i - stride] + out_ptr[i + 1 - stride] + out_ptr[i + 2 - stride] +
						out_ptr[i - 2] + out_ptr[i - 1] + out_ptr[i + 1] + out_ptr[i + 2]) > fiveby_filter_level - 6)
					{
						out_ptr[i] = 1;
					}
				}
			}
		}
	}
}

static __global__ void avg_out(float* out, hipComplex* d_fft, const int num_wins, const int averaging, const float offset, const int resolution) {
	
	//Need to modify for appropriate averaging output
	int idx = threadIdx.x;
	float* out_ptr = &out[0];
	hipComplex* d_fft_ptr = &d_fft[0];
	const float threshold = -96;

	bool THRESHOLD = true;

	for (int j = 0; j < num_wins; j++){ //what about the final set of frames? They should be retained and re-computed to maintain accurate averaging ...

		for (int i = blockIdx.x * blockDim.x + idx; i < resolution*averaging; i += blockDim.x * gridDim.x){
			//Moving average of each output bin according to the 'averaging' value - typically set to 10
			out_ptr[((resolution / 2) + i) % resolution] += (
				10 * log10(abs(d_fft_ptr[i].x * d_fft_ptr[i].x + d_fft_ptr[i].y * d_fft_ptr[i].y) / resolution) //DFT bin magnitude
				);
		}

//		__syncthreads();

		if (THRESHOLD){
			out_ptr[(resolution / 2 + blockIdx.x * blockDim.x + idx) % resolution] = ((out_ptr[(resolution / 2 + blockIdx.x * blockDim.x + idx) % resolution] / averaging + offset) <= threshold) ? 1 : 0;
		}
		else {
			out_ptr[(resolution / 2 + blockIdx.x * blockDim.x + idx) % resolution] = (out_ptr[(resolution / 2 + blockIdx.x * blockDim.x + idx) % resolution] / averaging + offset);
		}
//		if (out_ptr[blockIdx.x * blockDim.x + idx] <= threshold) out_ptr[blockIdx.x * blockDim.x + idx] = 1;
//		elseP out_ptr[blockIdx.x * blockDim.x + idx] = 0;

		out_ptr += resolution; //increment out_ptr by one frame of averages
		d_fft_ptr += resolution; //increment d_fft_ptr by one frame to maintain rolling average
	}
}

/* DEPRECATED
static __global__ void avg_out_filter(float* out, hipComplex* d_fft, const int num_wins, const int averaging, const float offset, const int resolution) {

	//Need to modify for appropriate averaging output
	//Remember whitespace is a 1
	int idx = threadIdx.x;
	float* out_ptr = &out[0];
	hipComplex* d_fft_ptr = &d_fft[0];
	const float threshold = -96;
	const int filter_level = 13; //normally 16 for fiveby, 13 for aggressive

	bool THRESHOLD = true;
	bool FILTER = true;
	bool FIVEBY = false;

	for (int j = 0; j < num_wins; j++){

		for (int i = blockIdx.x * blockDim.x + idx; i < resolution*averaging; i += blockDim.x * gridDim.x){

			out_ptr[((resolution / 2) + i) % resolution] += (
				10 * log10(abs(d_fft_ptr[i].x * d_fft_ptr[i].x + d_fft_ptr[i].y * d_fft_ptr[i].y) / resolution) //DFT bin magnitude
				);
		}

		//		__syncthreads();

		if (THRESHOLD){
			out_ptr[(resolution / 2 + blockIdx.x * blockDim.x + idx) % resolution] = ((out_ptr[(resolution / 2 + blockIdx.x * blockDim.x + idx) % resolution] / averaging + offset) <= threshold) ? 1 : 0;
		}
		else {
			out_ptr[(resolution / 2 + blockIdx.x * blockDim.x + idx) % resolution] = (out_ptr[(resolution / 2 + blockIdx.x * blockDim.x + idx) % resolution] / averaging + offset);
		}
		//		if (out_ptr[blockIdx.x * blockDim.x + idx] <= threshold) out_ptr[blockIdx.x * blockDim.x + idx] = 1;
		//		elseP out_ptr[blockIdx.x * blockDim.x + idx] = 0;

		out_ptr += resolution; //increment out_ptr by one frame of averages
		d_fft_ptr += resolution; //increment d_fft_ptr by number of frames averaged
	}

	//Now perform filtering, only if thresholding is performed

	if (THRESHOLD && FILTER && !FIVEBY){
		//Zero out pointer
		out_ptr = &out[0 + resolution]; //we dont want to filter the first row - at this stage anyway
		int absthreadidx = blockIdx.x * blockDim.x + threadIdx.x; //I wanted to be more explicit before, shortcutting here

		//j starts at 1 and ends at num_wins-1 to give the sufficient spacing for the 3x3 kernel
		for (int j = 1; j < num_wins-1; j++){

			if (j == 0) { //first row
			}
			else if (j == num_wins - 1) { //last row
			
			}

			if (absthreadidx == 0) { //left edge
			}
			else if (absthreadidx == resolution - 1) { //right edge
			}

			else { //everything else
				//If the centre of a kernel = 1, take a 3 by 3 kernel, and sum the edge cells, if greater than 7, can assume this is noise
				if (out_ptr[absthreadidx] == 0)
				{
					//Currently set to detect a lone cell. Can increase this for more agressive filtering. Though the kernel size may have to increase also
					if ((out_ptr[absthreadidx - resolution - 1] + out_ptr[absthreadidx - resolution] + out_ptr[absthreadidx - resolution + 1] +
						out_ptr[absthreadidx - 1] + out_ptr[absthreadidx + 1] +
						out_ptr[absthreadidx + resolution - 1] + out_ptr[absthreadidx + resolution] + out_ptr[absthreadidx + resolution + 1]) > 6)
					{
						out_ptr[absthreadidx] = 1;
					}
				}
			}

			out_ptr += resolution; //next row of output array (as the 2d output is really just a very long 1d array)
		}
	}

	else if (THRESHOLD && FILTER && FIVEBY){
		//Zero out pointer
		out_ptr = &out[0 + 2 * resolution]; //we dont want to filter the first 2 rows - at this stage anyway
		int absthreadidx = blockIdx.x * blockDim.x + threadIdx.x; //I wanted to be more explicit before, shortcutting here

		//j starts at 1 and ends at num_wins-1 to give the sufficient spacing for the 3x3 kernel
		for (int j = 2; j < num_wins - 2; j++){

			if (j == 0 || j == 1) { //first row
			}
			else if (j == num_wins - 2 || j == num_wins - 1) { //last row

			}

			if (absthreadidx == 0 || absthreadidx == 1) { //left edge
			}
			else if (absthreadidx == resolution - 1 || absthreadidx == resolution - 2) { //right edge
			}

			else { //everything else
				//If the centre of a kernel = 1, take a 3 by 3 kernel, and sum the edge cells, if greater than filter_level, can assume this is noise
				if (out_ptr[absthreadidx] == 0)
				{
					//Currently set to detect a lone cell. Can increase this for more agressive filtering. Though the kernel size may have to increase also
					if ((
						out_ptr[absthreadidx - 2 * resolution - 2] + out_ptr[absthreadidx - 2 * resolution - 1] + out_ptr[absthreadidx - 2 * resolution] + out_ptr[absthreadidx - 2 * resolution + 1] + out_ptr[absthreadidx - 2 * resolution + 2] +
						out_ptr[absthreadidx - resolution - 2] + out_ptr[absthreadidx - resolution - 1] + out_ptr[absthreadidx - resolution] + out_ptr[absthreadidx - resolution + 1] + out_ptr[absthreadidx - resolution + 2] +
						out_ptr[absthreadidx - 2] + out_ptr[absthreadidx - 1] + out_ptr[absthreadidx + 1] + out_ptr[absthreadidx + 2] +
						out_ptr[absthreadidx + resolution - 2] + out_ptr[absthreadidx + resolution - 1] + out_ptr[absthreadidx + resolution] + out_ptr[absthreadidx + resolution + 1] + out_ptr[absthreadidx + resolution + 2] +
						out_ptr[absthreadidx + 2 * resolution - 2] + out_ptr[absthreadidx + 2 * resolution - 1] + out_ptr[absthreadidx + 2 * resolution] + out_ptr[absthreadidx + 2 * resolution + 1] + out_ptr[absthreadidx + 2 * resolution + 2]
						) > filter_level)
					{
						out_ptr[absthreadidx] = 1;
					}
				}
			}

			out_ptr += resolution; //next row of output array (as the 2d output is really just a very long 1d array)
		}
	}

}
/*

/* BACKUP LOL
static __global__ void avg_out(float* out, hipComplex* d_fft, const int num_wins, const int averaging) {

	int idx = threadIdx.x;
	float* out_ptr = &out[0];
	hipComplex* d_fft_ptr = &d_fft[0];

	for (int j = 0; j < num_wins / averaging; j++){

		for (int i = blockIdx.x * blockDim.x + idx; i < NUM_SAMPS*averaging; i += blockDim.x * gridDim.x){

			out_ptr[i%NUM_SAMPS] += (
				10 * log10(abs(d_fft_ptr[i].x * d_fft_ptr[i].x + d_fft_ptr[i].y * d_fft_ptr[i].y) / NUM_SAMPS) //DFT bin magnitude
				);
		}
		out_ptr += NUM_SAMPS; //increment out_ptr by one frame of averages
		d_fft_ptr += NUM_SAMPS*averaging; //increment d_fft_ptr by number of frames averaged
	}
}*/

void cuda_memcheck() {
	size_t free_byte;

	size_t total_byte;

	hipError_t cudaStatus;

	cudaStatus = hipMemGetInfo(&free_byte, &total_byte);

	size_t used_byte = total_byte - free_byte;

	if (cudaStatus != hipSuccess){

		printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cudaStatus));

		exit(1);

	}
	else printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",

		used_byte / 1024.0 / 1024.0, free_byte / 1024.0 / 1024.0, total_byte / 1024.0 / 1024.0);
}